#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define epsilon 0.00001
#define maxRep 40

__global__ void meanShift(double *x,size_t pitchx,double *y, size_t pitchy,double *ynew,size_t pitchynew,int N,int d,double sigma);
__device__ double calcDist(double *y,size_t pitchy,double *x,size_t pitchx,int d);
__device__ double gausK(double x,double sigma);
double froNorm(double *a,size_t pitcha,double *b,size_t pitchb,int N,int d);
void test(double *y,size_t pitchy,char *testfile,int N,int d);

int main(int argc,char **argv){	

	if(argc!=4){
		printf("Usage: %s (dataset) (test) (sigma) where (dataset) ",argv[0]);
		printf("is the name of the dataset .txt file, (test) is the name of the ");
		printf(".txt test file and (sigma) is the value of sigma for the current dataset\n");
		exit(1);
	}

	struct timeval startwtime, endwtime;
	double time;

	//turn (sigma) input from string to double
	double sigma=atof(argv[3]);

	int i,j; 

	//argv[1] is the (dataset) file
	FILE *file = fopen(argv[1], "r");
	if(file==NULL){
		printf("Couldn't open %s\n",argv[1]);
		exit(1);
	}

	//count the number of points and dimensions of (dataset)
	int d=0,N=0;
	char ch;

	/**dimension and number of points counting found in
	 *https://www.opentechguides.com/how-to/article/c/72/c-file-counts.html
	*/
	while ((ch=getc(file)) != EOF) {
		if ((ch == ' ' || ch == '\n') && N==0) { ++d; }
		
		if (ch == '\n') { ++N; }
	}

	//1 dimension host memory allocation to fit cudaMemcpy2D
	double *y;
	size_t pitchy = sizeof(double) * d;
	y = (double*)malloc(sizeof(double) * N * d);

	double *ynew;
	size_t pitchynew = sizeof(double) * d;
	ynew = (double*)malloc(sizeof(double) * N * d);

	double *x;
	size_t pitchx = sizeof(double) * d;
	x = (double*)malloc(sizeof(double) * N * d);

	double *row_x,*row_y; 
	//return file pointer to the beggining of the file
	fseek(file, 0, SEEK_SET);
	for (i=0;i<N;i++){
		row_x = (double*)((char*)x + i * pitchx );
		row_y = (double*)((char*)y + i * pitchy  );
		for (j=0;j<d;j++){
			fscanf(file,"%lf",&row_x[j]);
			row_y[j]=row_x[j];
		}

	}

	fclose(file);

	//allocate 2d arrays for device memory
	double *d_x;
	double *d_y;
	double *d_ynew;
	size_t d_pitchx,d_pitchy,d_pitchynew;

	hipMallocPitch((void**)&d_x, &d_pitchx, d * sizeof(double), N);
	hipMallocPitch((void**)&d_y, &d_pitchy, d * sizeof(double), N);
	hipMallocPitch((void**)&d_ynew, &d_pitchynew, d * sizeof(double), N);

	//copy data from host to device memory
	hipMemcpy2D(d_x,d_pitchx,x,pitchx, d * sizeof(double), N, hipMemcpyHostToDevice);
	hipMemcpy2D(d_y,d_pitchy,y,pitchy, d * sizeof(double), N, hipMemcpyHostToDevice);




	int repeats=0;
	double norm;
	double *row_ynew;
	gettimeofday (&startwtime, NULL);

	do{
		meanShift<<<N,d>>>(d_x,d_pitchx,d_y,d_pitchy,d_ynew,d_pitchynew,N,d,sigma);
		hipMemcpy2D(y, sizeof(double)*d, d_y, d_pitchy, sizeof(double) * d, N, hipMemcpyDeviceToHost);
		
		//calculate norm of (ynew-y)
		norm = froNorm(y,pitchy,ynew,pitchynew,N,d);
		
		//update ynew after a meanshift iteration
		for (i=0;i<N ;i++){
			row_ynew = (double*)((char*)ynew + i * pitchynew);
			row_y = (double*)((char*)y +i * pitchy);
			for (j=0;j<d;j++){
				row_ynew[j] = row_y[j];
			}
		}
		repeats++;

	}while(norm>epsilon && repeats<maxRep);

	gettimeofday (&endwtime, NULL); 
	
	time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
	+ endwtime.tv_sec - startwtime.tv_sec);
		
	printf("Wall clock time: %f \n", time);

	//argv[2] is the (testfile) name
	test(y,pitchy,argv[2],N,d);

	return 0;

}

__global__ void meanShift(double *x,size_t pitchx,double *y, size_t pitchy,double *ynew,size_t pitchynew,int N,int d,double sigma){
	int index=blockDim.x*blockIdx.x+threadIdx.x;

	if (index<N){
		double sum=0,res=0;
		int j,k;
		
		double* row_y=(double*)((char*)y+index*pitchy);	
		double* row_ynew=(double*)((char*)ynew+index*pitchynew);
		
		//initialize ynew
		for(k=0;k<d;k++)
			row_ynew[k]=0;
				
		for(j=0;j<N;j++){
			double* row_x=(double*)((char*)x+j*pitchx);
			
			if(calcDist(row_y,pitchy,row_x,pitchx,d)<sigma*sigma){
				double temp=0;
				for(k=0;k<d;k++){
					temp=(row_y[k]-row_x[k])*(row_y[k]-row_x[k])+temp;
					//temp is the square of norm2(y_i-x_j)
				}	
				res=gausK(temp,sigma);
				
				for(k=0;k<d;k++){
					row_ynew[k]=row_ynew[k]+row_x[k]*res;
				}
				sum=sum+res;
				//calculating denominator of ynew_i
			}	
		}
		for(k=0;k<d;k++){
				row_ynew[k]=row_ynew[k]/sum;
		}
		//update y from all threads
		for(k=0;k<d;k++){
				row_y[k]=row_ynew[k];
		}
	}
}

//calculate distance between x and y
__device__ double calcDist(double *y,size_t pitchy,double *x,size_t pitchx,int d){
	double sum = 0;
	int l;

	for (l=0;l<d;l++){
		sum = sum + (y[l]-x[l])*(y[l]-x[l]);
	}

	return sqrt(sum);
}

__device__ double gausK(double x,double sigma){
	double f;
	f = exp(-x/(2*(sigma*sigma)));

	return f;
}

//calculate frobenius norm of (a-b)
double froNorm(double *a,size_t pitcha,double *b,size_t pitchb,int N,int d){
	int i,j;
	double sum=0;
	double *row_b,*row_a;

	for (i=0;i<N;i++){
		row_a = (double*)((char*)a + i * pitcha);
		row_b = (double*)((char*)b + i * pitchb);
		for (j=0;j<d;j++){
			sum = sum + (row_a[j]-row_b[j])*(row_a[j]-row_b[j]);
		}
	}

	return sqrt(sum);
}

void test(double *y,size_t pitchy,char *testfile,int N,int d){
	int i,j;
	double **test;

	//memory allocation for test input
	test =(double **) malloc(sizeof(double*)*N);

	for (i=0;i<N;i++){
		test[i] = (double *)malloc(sizeof(double)*d);
	}
	FILE *file = fopen(testfile, "r"); 
	if(file==NULL){
		printf("Couldn't open %s\n",testfile);
		exit(1);
	}
	
	for (i=0;i<N;i++){
		for (j=0;j<d;j++){
			fscanf(file,"%lf",&test[i][j]);
		}
	}

	//compare the arrays
	int failed=0;
	for (i=0;i<N;i++){
		double* row_y=(double*)((char*)y+i*pitchy);	
		for (j=0;j<d;j++){
			//check if relative error to matlab output is small
			if (fabs(row_y[j]-(double)test[i][j])/fabs((double)test[i][j]) > 0.1)
				failed++;
		}
	}

	//check if a small percentage of the result is wrong
	if((double)(d*N-failed)/(double)(d*N)*100<95.0)
		printf("Test failed!\n");
	else
		printf("Test passed!\n");

	fclose(file);
}

